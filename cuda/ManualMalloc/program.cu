#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define count 11 //giju skaicius
#define masyvuCount 5	//masyvu skaicius

struct Item
{
	char name[20 * count];
	int amount;
	int eilNr; // eiles numeris masyve
	double price;
};

struct Supplier {
	struct Item items[count];
	int itemCount;
	char supplierName[20];
};

//tiekeju masyvas
struct Supplier suppliers[masyvuCount];

//bendras masyvas
struct Item bendras[count];

void readFile();
void printFile(struct Item supplier[], int  itemCount);
void printSuppliers(struct Supplier suppliers[]);
void printItemsArray(struct Item supplier[], int itemCount);
void printItemsResults(struct Item supplier[], int  itemCount);

__device__ void addElements(struct Item bendras[], struct Supplier duom[], int id);
__device__ void addElement(struct Item bendras[], struct Item element, int index);
__device__ char * cuda_strcpy(char *dest, const char *src);
__device__ char * cuda_strcat(char *dest, const char *src);

/** Pagrindine lygiagreti funkcija
*	@param f - bendras tiekeju masyvas
*	@param bendras - bendras masyvas
*/
__global__ void addKernel(struct Supplier f[], struct Item bendras[])
{
	int tid = threadIdx.x; 			//gijos numeris
	addElements(bendras, f, tid);	//elemento sudejimas i bendra masyva
}

/**Elemento sudejimo funkcija i bendra masyva
*	@param bendras - bendras masyvas
*	@param duom - tiekeju masyvas
*	@param id - gijos numeris, pagal kuria imsim duomenis
*/
__device__ void addElements(struct Item bendras[], struct Supplier duom[], int id) {
	for (int j = 0; j < duom[0].itemCount; j++) {
		addElement(bendras, duom[j].items[id], id);
	}
}

/**Vieno elemento pridejimas
*	@param bendras	- bendras masyvas
*	@param element - preke
*	@param index - bendro masyvo indeksas
*/
__device__ void addElement(struct Item bendras[], struct Item element, int index) {
	bendras[index].eilNr = index;
	bendras[index].price = bendras[index].price + element.price;
	bendras[index].amount = bendras[index].amount + element.amount;
	cuda_strcat(bendras[index].name, element.name);
}

/** Stringo kopijavimas
*	@param dest - kur talpinsime
*	@param src - ka talpinsime
*/
__device__ char * cuda_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

/*	Stringo kopijavimas
*	@param dest - kur talpinsime
*	@param src - ka talpinsime
*/
__device__ char * cuda_strcat(char *dest, const char *src) {
	int i = 0;
	while (dest[i] != 0) i++;
	cuda_strcpy(dest + i, src);
	return dest;
}

int main(int argc, char *argv[])
{
	readFile();
	// pradiniu duomenu spausdinimas
	printSuppliers(suppliers);
	// tiekejai
	struct Supplier *s;
	// bendras
	struct Item *bendrasCuda;

	//atminties isskyrimas GPU
	hipMalloc((void**)&s, sizeof(Supplier)*masyvuCount);
	// kopijavimas i GPU
	hipMemcpy(s, suppliers, sizeof(Supplier)*masyvuCount, hipMemcpyHostToDevice);
	//atminities isskyrimas i GPU
	hipMalloc((void**)&bendrasCuda, sizeof(Item)*count);
	//kopijavimas i GPU
	hipMemcpy(bendrasCuda, bendras, sizeof(Item)*count, hipMemcpyHostToDevice);

	// giju skaiciaus parinkimas ir lygiagretaus kodo startavimas
	addKernel << < 1, count >> >(s, bendrasCuda);
	
	// kopijavimas i CPU
	hipMemcpy(bendras, bendrasCuda, sizeof(Item)*count, hipMemcpyDeviceToHost);
	//atlaisvinimas atminties is GPU	
	hipFree((void**)&s);
	//atlaisvinimas atminties is GPU
	hipFree((void**)&bendrasCuda);

	printItemsResults(bendras, count);
	printFile(bendras, count);
	printf("Done \n");
	return 0;
}


/**
*	@param suppliers - tiekeju masyvas
*/
void printSuppliers(struct Supplier suppliers[]) {
	for (int i = 0; i < masyvuCount; i++)
	{
		printf("*** %s *** \n", suppliers[i].supplierName);
		printf("  %-10s %-10s %-4s \n", "Name", "Amount", "Price");
		printItemsArray(suppliers[i].items, suppliers[i].itemCount);
	}
}

void printItemsArray(struct Item supplier[], int  itemCount) {
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			printf("%d %-10s %-10d %-4.2f \n", supplier[i].eilNr, 
				supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}
}


/**
* Rezultatu spausdinimas
* @param supplier - tiekejo masyvas
*/
void printItemsResults(struct Item supplier[], int  itemCount) {
	printf("\nREZULTATAI \n\n");
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			printf("%d %-50s %-10d %-4.2f \n", supplier[i].eilNr, supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}
}

/**
*	Skaitymas is failo
*/
void readFile() {
	errno_t err;
	FILE *stream;
	char file_name[21] = "SankauskasS_L4.txt";  //failo vardas
	err = fopen_s(&stream, file_name, "r");
	
	char name[20];
	int n; //prekiu skaicius
	int supplierCount = 0; //tiekejo iteracijos kintamasis
	
	while (true) {
		int readItems = fscanf(stream, "%s %d", name, &n);
		if (readItems == 2) {
			strcpy(suppliers[supplierCount].supplierName, name);
			suppliers[supplierCount].itemCount = n;
			for (int i = 0; i < n; i++) {
				struct Item item = suppliers[supplierCount].items[i];
				char item_name[20];
				int amount;
				double price;
				fscanf(stream, "%s %d %lf", item_name, &amount, &price);
				suppliers[supplierCount].items[i].amount = amount;
				suppliers[supplierCount].items[i].eilNr = i;
				suppliers[supplierCount].items[i].price = price;
				strcpy(suppliers[supplierCount].items[i].name, item_name);
			}
			supplierCount++;
		}
		else if (readItems == EOF) {
			break;
		}
	}
	if (stream)
		err = fclose(stream);
}

/**
 * Rasymas i faila
 */
void printFile(struct Item supplier[], int  itemCount)
{
	FILE *f = fopen("SankauskasS_L4a_rez.txt", "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f,"\nREZULTATAI \n\n");
	fprintf(f, "%-3s %-50s %-10s %-7s \n", "Nr", "Name", "Amount", "Price");
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			fprintf(f, "%-3d %-50s %-10d %-7.2f \n", supplier[i].eilNr, 
				supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}

	fclose(f);
}